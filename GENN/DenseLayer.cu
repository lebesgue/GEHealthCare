#include "hip/hip_runtime.h"
#include "DenseLayer.cuh"
#include <stdexcept>

namespace cuda {


float __device__ get(Matrix A, int row, int col) {
	return A.data[row * A.w + col];
}


float __device__ get(Vector v, int pos) {
	return v.data[pos];
}


void __device__ set(Matrix A, int row, int col, float val) {
	A.data[row * A.w + col] = val;
}


void __device__ inc(Matrix A, int row, int col, float val) {
	A.data[row * A.w + col] += val;
}


void __global__ activate(float * x, int n, int activation) {
	int idx = threadIdx.x;
	if (idx >= n)
		return;

	if (activation == 0)
		x[idx] = 1.0f / (1.0f + expf(-x[idx]));
	if (activation == 1) {
		float sm = 0.0f;
		atomicAdd(&sm, expf(x[idx]));
		__syncthreads();
		x[idx] /= sm;
	}
}


void __global__ linear(DenseLayer layer) {
	extern __shared__ float sharedX[];

	int row = blockIdx.x;
	int col = threadIdx.x;

	if (row >= layer.W.h)
		return;
	if (col >= layer.W.w)
		return;

	sharedX[col] = get(layer.input, col);
	__syncthreads();

	__shared__ float val;
	if (col == 0)
		val = layer.b.data[row];
	__syncthreads();

	atomicAdd(&val, get(layer.W, row, col) * sharedX[col]);
	__syncthreads();

	if (col == 0)
		layer.output.data[row] = val;
}


__global__ void grad(DenseLayer layer) {
	int row = blockIdx.x;
	int col = threadIdx.x;

	if (row >= layer.gradW.h || col >= layer.gradW.w)
		return;

	__shared__ float gradAct;
	if (col == 0) {
		if (layer.activation == 0) {
			gradAct = layer.output.data[row] * (1 - layer.output.data[row]);
		}
		else {
			gradAct = 1.0f;
		}
	}
	__syncthreads();

	inc(layer.gradW, row, col, layer.input.data[col] * layer.dOutput.data[row] * gradAct);
	if (col == 0)
		layer.gradb.data[row] += gradAct * layer.dOutput.data[row];

}


__global__ void backPropagate(DenseLayer layer) {
	int row = threadIdx.x;
	int col = blockIdx.x;

	if (row >= layer.out || col >= layer.in)
		return;

	extern __shared__ float sharedX[];
	sharedX[row] = get(layer.W, row, col);
	if (layer.activation == 0) {
		sharedX[row + blockDim.x] = layer.output.data[row] * (1 - layer.output.data[row]);
	}
	else {
		sharedX[row + blockDim.x] = 1.0f;
	}
	__syncthreads();

	__shared__ float di;
	if (row == 0)
		di = 0.0f;
	__syncthreads();

	atomicAdd(&di, sharedX[row] * sharedX[row + blockDim.x] * layer.dOutput.data[row]);
	__syncthreads();

	if (row == 0)
		layer.dInput.data[col] = di;
}


__global__ void stepKernel(DenseLayer layer, float learningRate) {
	int row = blockIdx.x;
	int col = threadIdx.x;

	if (row >= layer.out || col >= layer.in)
		return;
	__syncthreads();

	inc(layer.W, row, col, -get(layer.gradW, row, col) * learningRate);
	if (col == 0)
		layer.b.data[row] -= layer.gradb.data[row] * learningRate;
}


inline hipError_t checkCuda(hipError_t result) {
	if (result != hipSuccess) {
		throw std::runtime_error("CUDA Error");
	}
	return result;
}


void toGpu(float** dst, float** src, int s) {
	checkCuda(hipMalloc(dst, s * sizeof(float)));
	checkCuda(hipMemcpy(*dst, *src, s * sizeof(float), hipMemcpyHostToDevice));
}


void fromGpu(float** dst, float** src, int s) {
	checkCuda(hipMemcpy(*dst, *src, s * sizeof(float), hipMemcpyDeviceToHost));
}


void DenseLayer::initLayer() {
	W.w = in;
	W.h = out;
	b.s = out;

	gradW.w = in;
	gradW.h = out;
	gradb.s = out;

	input.s = in;
	dInput.s = in;
	output.s = out;
	dOutput.s = out;

	checkCuda(hipMalloc(&W.data, W.h * W.w * sizeof(float)));
	checkCuda(hipMalloc(&gradW.data, gradW.h * gradW.w * sizeof(float)));
	checkCuda(hipMalloc(&b.data, b.s * sizeof(float)));
	checkCuda(hipMalloc(&gradb.data, gradb.s * sizeof(float)));
	checkCuda(hipMemset(gradW.data, 0, gradW.h * gradW.w * sizeof(float)));
	checkCuda(hipMemset(gradb.data, 0, gradb.s * sizeof(float)));
	
	checkCuda(hipMalloc(&input.data, in * sizeof(float)));
	checkCuda(hipMalloc(&dInput.data, in * sizeof(float)));
	checkCuda(hipMalloc(&output.data, out * sizeof(float)));
	checkCuda(hipMalloc(&dOutput.data, out * sizeof(float)));
}


void DenseLayer::destroyLayer() {
	checkCuda(hipFree(&W.data));
	checkCuda(hipFree(&gradW.data));
	checkCuda(hipFree(&b.data));
	checkCuda(hipFree(&gradb.data));
	checkCuda(hipFree(&input.data));
	checkCuda(hipFree(&dInput.data));
	checkCuda(hipFree(&output.data));
	checkCuda(hipFree(&dOutput.data));
}


void DenseLayer::forward() {
	int sharedMemSize = in * sizeof(float) * 2;
	int rows = out;
	int cols = in;

	linear<<<rows, cols, sharedMemSize>>>(*this);
	activate<<<1, rows>>>(output.data, out, activation);
	hipDeviceSynchronize();
}


void DenseLayer::backward() {
	int sharedMemSize = out * sizeof(float) * 4;
	int rows = out;
	int cols = in;

	grad<<<rows, cols, sharedMemSize>>>(*this);
	backPropagate<<<cols, rows, sharedMemSize>>>(*this);
	hipDeviceSynchronize();
}


void DenseLayer::step(float learningRate) {
	int rows = out;
	int cols = in;
	stepKernel<<<rows, cols>>>(*this, learningRate);
	hipDeviceSynchronize();
}


void DenseLayer::zeroGrad() {
	checkCuda(hipMemset(gradW.data, 0, gradW.h * gradW.w * sizeof(float)));
	checkCuda(hipMemset(gradb.data, 0, gradb.s * sizeof(float)));
	hipDeviceSynchronize();
}


void DenseLayer::initBackProp(int label) {
	if (!isOutput)
		return;

	float* o = (float*)malloc(out * sizeof(float));
	fromGpu(&o, &output.data, out);

	float sum = 0.0;
	for (int i = 0; i < out; i++) {
		o[i] = expf(o[i]);
		sum += o[i];
	}

	for (int i = 0; i < out; i++) {
		o[i] /= sum;
		o[i] -= i == label ? 1.0f : 0.0f;
	}
	toGpu(&dOutput.data, &o, out);
}


int DenseLayer::argmax() {
	float* o = (float*)malloc(out * sizeof(float));
	fromGpu(&o, &output.data, out);
	float m = o[0];
	int im = 0;
	for (int i = 1; i < out; i++) {
		if (o[i] > m) {
			im = i;
			m = o[i];
		}
	}
	return im;
}


float DenseLayer::loss(int label) {
	float denom = 0.0f;
	float * sm = (float *)malloc(out * sizeof(float));
	fromGpu(&sm, &output.data, out);
	for (int i = 0; i < out; i++) {
		sm[i] = expf(sm[i]);
		denom += sm[i];
	}
	return -logf(sm[label] / denom);
}


}
